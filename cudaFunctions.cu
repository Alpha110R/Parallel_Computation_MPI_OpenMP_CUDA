#include "hip/hip_runtime.h"
#if 0
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#endif

#include <stdio.h>
#include "functions.h"
#define THREADS_PER_BLOCK 256
#define HISTOGRAMA_SIZE 256

__global__  void histogramaCuda(int *d_Array, int quarterOfFullSize, int* d_Histograma) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    __shared__ int sharedArrayHistogram[HISTOGRAMA_SIZE];
     sharedArrayHistogram[threadIdx.x] = 0;
    __syncthreads();

    if (id < quarterOfFullSize)
        atomicAdd(&(sharedArrayHistogram[d_Array[id]]), 1);
    __syncthreads();
    atomicAdd(&(d_Histograma[threadIdx.x]), sharedArrayHistogram[threadIdx.x]);
}

// returns 1 if successful, otherwise returns 0
int computeOnGPU(int *arrayOfNumbers, int quarterOfFullSize, int* histograma) {
    //Calculate the second half of the slave array
    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;
    int cudaHistograma[HISTOGRAMA_SIZE]= {0};
    int* d_Array;
    int* d_Histograma;
    err = hipMalloc((void **)&d_Array, quarterOfFullSize * sizeof(int));
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to allocate device memory - %s\n", hipGetErrorString(err));
        return 0;
    }
    err = hipMalloc((void **)&d_Histograma, HISTOGRAMA_SIZE * sizeof(int));
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to allocate device memory - %s\n", hipGetErrorString(err));
        return 0;
    }
    hipMemset(d_Histograma, 0, HISTOGRAMA_SIZE * sizeof(int));

    // Copy data from host to the GPU memory
    err = hipMemcpy(d_Array, arrayOfNumbers, quarterOfFullSize * sizeof(int), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to copy data from host to device - %s\n", hipGetErrorString(err));
        return 0;
    }

    // Launch the Kernel
    int blocksPerGrid =(quarterOfFullSize + THREADS_PER_BLOCK) / THREADS_PER_BLOCK;
    histogramaCuda<<<blocksPerGrid, THREADS_PER_BLOCK>>>(d_Array, quarterOfFullSize, d_Histograma);
    err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to launch incrementByOne kernel -  %s\n", hipGetErrorString(err));
        return 0;
    }

    // Copy the  result from GPU to the host memory.
    err = hipMemcpy(cudaHistograma, d_Histograma, HISTOGRAMA_SIZE * sizeof(int), hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to copy result array from device to host -%s\n", hipGetErrorString(err));
        return 0;
    }
    mergeHistograms(histograma, cudaHistograma);
    // Free allocated memory on GPU
    if (hipFree(d_Array) != hipSuccess) {
        fprintf(stderr, "Failed to free device data - %s\n", hipGetErrorString(err));
        return 0;
    }
    if (hipFree(d_Histograma) != hipSuccess) {
        fprintf(stderr, "Failed to free device data - %s\n", hipGetErrorString(err));
        return 0;
    }

    return 1;
}

