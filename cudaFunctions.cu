#include "hip/hip_runtime.h"
#if 0
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#endif

#include <stdio.h>
#include "functions.h"
#define THREADS_PER_BLOCK 256
__global__  void histograma(int *d_Aarray, int size, int* histograma) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    int histogramSize = size*2*2;
    __shared__ int sharedArrayHistogram [histogramSize];
     sharedArrayHistogram[threadIdx.x] = 0;
    __syncthreads();

    if (id < size)
        atomicAdd(&(sharedArrayHistogram[d_Aarray[id]]), 1);
    __syncthreads();

    atomicAdd(&histograma[threadIdx.x], sharedArrayHistogram[threadIdx.x]);

}

int* createsTheSecondHalfOfArray(int* array, int fullSize){
    int size = fullSize/2;
    int* secondHalfArray= (int*)malloc(size * sizeof(int));
    for(int i=size; i<fullSize ; i++){
        secondHalfArray[i] = array[i];
    }
    return secondHalfArray;
}

// returns 1 if successful, otherwise returns 0
int computeOnGPU(int *arrayOfNumbers, int amountOfNumbersToSlave, int* histograma) {
    //Culcolate the second half of the slave array
    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;

    int size = (amountOfNumbersToSlave/2) * sizeof(int);//size_t
    int* newArrayOfNumbers = createsTheSecondHalfOfArray(arrayOfNumbers, amountOfNumbersToSlave);//the size is size

    // Allocate memory on GPU to copy the data from the host
    int* d_Aarray;
    int* d_Histograma;
    err = hipMalloc((void **)&d_Aarray, size);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to allocate device memory - %s\n", hipGetErrorString(err));
        return 0;
    }
    err = hipMalloc((void **)&d_Histograma, amountOfNumbersToSlave*2);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to allocate device memory - %s\n", hipGetErrorString(err));
        return 0;
    }

    // Copy data from host to the GPU memory
    err = hipMemcpy(d_Aarray, newArrayOfNumbers, size, hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to copy data from host to device - %s\n", hipGetErrorString(err));
        return 0;
    }
    err = hipMemcpy(d_Histograma, histograma, amountOfNumbersToSlave*2, hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to copy data from host to device - %s\n", hipGetErrorString(err));
        return 0;
    }


    // Launch the Kernel
    int blocksPerGrid =(size + THREADS_PER_BLOCK) / THREADS_PER_BLOCK;
    histograma<<<blocksPerGrid, THREADS_PER_BLOCK>>>(d_Aarray, size, histograma);
    err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to launch incrementByOne kernel -  %s\n", hipGetErrorString(err));
        return 0;
    }

    // Copy the  result from GPU to the host memory.
    err = hipMemcpy(histograma, d_Histograma, amountOfNumbersToSlave*2, hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to copy result array from device to host -%s\n", hipGetErrorString(err));
        return 0;
    }

    // Free allocated memory on GPU
    if (hipFree(d_Aarray) != hipSuccess) {
        fprintf(stderr, "Failed to free device data - %s\n", hipGetErrorString(err));
        return 0;
    }

    return 1;
}

