#include "hip/hip_runtime.h"
#if 0
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#endif

#include <stdio.h>
#include "functions.h"
#define THREADS_PER_BLOCK 256
#define HISTOGRAMA_SIZE 256

__global__  void histogramaCuda(int *d_Array, int quarterOfFullSize, int* d_Histograma) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    __shared__ int sharedArrayHistogram[HISTOGRAMA_SIZE];
     sharedArrayHistogram[threadIdx.x] = 0;
    __syncthreads();

    if (id < quarterOfFullSize)
        atomicAdd(&(sharedArrayHistogram[d_Array[id]]), 1);
    __syncthreads();
    atomicAdd(&(d_Histograma[threadIdx.x]), sharedArrayHistogram[threadIdx.x]);
}

// returns 1 if successful, otherwise returns 0
int computeOnGPU(int *arrayOfNumbers, int quarterOfFullSize, int* histograma, int* cudaHistograma) {
    //Culcolate the second half of the slave array
    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;

    // Allocate memory on GPU to copy the data from the host
    int* d_Aarray;
    int* d_Histograma;
    err = hipMalloc((void **)&d_Aarray, quarterOfFullSize * sizeof(int));
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to allocate device memory - %s\n", hipGetErrorString(err));
        return 0;
    }
    err = hipMalloc((void **)&d_Histograma, HISTOGRAMA_SIZE * sizeof(int));
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to allocate device memory - %s\n", hipGetErrorString(err));
        return 0;
    }
    hipMemset(d_Histograma, 0, HISTOGRAMA_SIZE * sizeof(int));

    // Copy data from host to the GPU memory
    err = hipMemcpy(d_Aarray, arrayOfNumbers, quarterOfFullSize * sizeof(int), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to copy data from host to device - %s\n", hipGetErrorString(err));
        return 0;
    }

    // Launch the Kernel
    int blocksPerGrid =(quarterOfFullSize + THREADS_PER_BLOCK) / THREADS_PER_BLOCK;
    histogramaCuda<<<blocksPerGrid, THREADS_PER_BLOCK>>>(d_Aarray, quarterOfFullSize, d_Histograma);
    err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to launch incrementByOne kernel -  %s\n", hipGetErrorString(err));
        return 0;
    }

    // Copy the  result from GPU to the host memory.
    err = hipMemcpy(cudaHistograma, d_Histograma, HISTOGRAMA_SIZE * sizeof(int), hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to copy result array from device to host -%s\n", hipGetErrorString(err));
        return 0;
    }
    mergeHistograms(histograma, cudaHistograma);
    // Free allocated memory on GPU
    if (hipFree(d_Aarray) != hipSuccess) {
        fprintf(stderr, "Failed to free device data - %s\n", hipGetErrorString(err));
        return 0;
    }
    if (hipFree(d_Histograma) != hipSuccess) {
        fprintf(stderr, "Failed to free device data - %s\n", hipGetErrorString(err));
        return 0;
    }

    return 1;
}

