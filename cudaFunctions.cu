#include "hip/hip_runtime.h"
#if 0
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#endif

#include <stdio.h>
#include "functions.h"
#define THREADS_PER_BLOCK 256
#define HISTOGRAMA_SIZE 256

__global__  void histogramaCuda(int *d_Aarray, int size, int* d_Histograma) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    __shared__ int sharedArrayHistogram[HISTOGRAMA_SIZE];
     sharedArrayHistogram[threadIdx.x] = 0;
     for(int i=0; i< size; i++){
      d_Histograma[i] = 0;
     }
    __syncthreads();

    if (id < size)
        atomicAdd(&(sharedArrayHistogram[d_Aarray[id]]), 1);
    __syncthreads();

    atomicAdd(&d_Histograma[threadIdx.x], sharedArrayHistogram[threadIdx.x]);
    if(id==0){
        printf("%d\n", d_Histograma[250]);
        printf("%d\n", sharedArrayHistogram[250]);

    }
}

// returns 1 if successful, otherwise returns 0
int computeOnGPU(int *arrayOfNumbers, int amountOfNumbersToSlave, int* histograma) {
    //Culcolate the second half of the slave array
    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;

    int size = (amountOfNumbersToSlave/2);//size_t
    //int* newArrayOfNumbers = createsTheSecondHalfOfArray(arrayOfNumbers, amountOfNumbersToSlave);//the size is size
    int HistogramaFromCuda[HISTOGRAMA_SIZE];
    initializeArray(HistogramaFromCuda, HISTOGRAMA_SIZE);
    // Allocate memory on GPU to copy the data from the host
    int* d_Aarray;
    int* d_Histograma;
    err = hipMalloc((void **)&d_Aarray, size);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to allocate device memory - %s\n", hipGetErrorString(err));
        return 0;
    }
    err = hipMalloc((void **)&d_Histograma, HISTOGRAMA_SIZE);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to allocate device memory - %s\n", hipGetErrorString(err));
        return 0;
    }

    // Copy data from host to the GPU memory
    err = hipMemcpy(d_Aarray, arrayOfNumbers, size, hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to copy data from host to device - %s\n", hipGetErrorString(err));
        return 0;
    }

    // Launch the Kernel
    int blocksPerGrid =(size + THREADS_PER_BLOCK) / THREADS_PER_BLOCK;
    histogramaCuda<<<blocksPerGrid, THREADS_PER_BLOCK>>>(d_Aarray, size, d_Histograma);
    err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to launch incrementByOne kernel -  %s\n", hipGetErrorString(err));
        return 0;
    }
    //printf("%d\n", d_Histograma[250]);
    // Copy the  result from GPU to the host memory.
    err = hipMemcpy(HistogramaFromCuda, d_Histograma, HISTOGRAMA_SIZE, hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to copy result array from device to host -%s\n", hipGetErrorString(err));
        return 0;
    }

    // Free allocated memory on GPU
    if (hipFree(d_Aarray) != hipSuccess) {
        fprintf(stderr, "Failed to free device data - %s\n", hipGetErrorString(err));
        return 0;
    }
    if (hipFree(d_Histograma) != hipSuccess) {
        fprintf(stderr, "Failed to free device data - %s\n", hipGetErrorString(err));
        return 0;
    }

    return 1;
}

